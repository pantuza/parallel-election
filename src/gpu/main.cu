#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <time.h>
#include "words.h"
#include "sentiment_analyser.h"
#include "tweet_target_analyser.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#define BLOCKSIZE	1048576


/* CUDA */
#define N (2048 * 2048)
#define THREADS_PER_BLOCK 1024;


using namespace std;


__global__ void kernel(words *d_dict, int Ncandidates, words **candidate_tags, int Ntweet, tweet *t, int *result)
{

	__shared__ words candidates[THREADS_PER_BLOCK];
	(words**) candidates = candidates_tags;
	 
	int j;

	__shared__ int sentiments[THREADS_PER_BLOCK];
	int index  = threadIdx.x + blockIdx.x * blockDim.x;

    sentiments[threadIdx.x] = analyse(dict, t[index].t);

	if (sentiments[threadIdx.x])
	{
	
		/* Every thread waits here to compute its sentiment by candidate */
		__syncthreads();
		

		/* Still can be parallelized */
		for (j = 0; j < Ncandidates; j++)
		{
			if (analyse(candidates[j], t[index].t))
			{

				if (sentiments[threadIdx.x] > 0)
					result[j * 2]++;
				else
					result[j * 2+1]++;
			}
		}
	}
}



FILE *openfile(const char * filename, const char * mode)
{
	FILE *f;
	f = fopen( filename,  mode);
	if (!f)cout<<endl << "Failed to load file" << filename << endl;
	return f;
}

int main(int argc, char *argv[])
{

	
	


	FILE *in;
	int i,j,Ntweets, Ntargets,*result;
	words *dict,**candidate_tag;
	tweet *t;
	float timestart;
	
	timestart=(float)clock()/CLOCKS_PER_SEC;
	if (argc < 3)
	{
		cout <<endl<< "expected input files:"<<endl<<"./exec <tweets> <dictionary> <target_dic1> <target_dic2> ..." << endl;
		return-1;
	}
//Load dictionary
	in = openfile(argv[2], "r");
	if (!in)return EXIT_FAILURE;
	dict = new words(count_words(in));

	/* CUDA dict */ 
	words *d_dict;
	hipMalloc(&d_dict, sizeof(dict));

	

	dict->fload_words(in);
	fclose(in);
//Load targets tags
	Ntargets = argc - 3;
	candidate_tag = new words*[Ntargets];
	for (i = 0; i < Ntargets; i++)
	{
		in = openfile(argv[i+3], "r");
		if (!in)return EXIT_FAILURE;
		candidate_tag[i] = new words(count_words(in));
		candidate_tag[i]->fload_words(in);
		fclose(in);
	}
	in = openfile(argv[1], "r");
	if (!in)return EXIT_FAILURE;
	Ntweets = BLOCKSIZE;
//Create resul array
	result = new int[Ntargets*2];
	for(i=0;i<Ntargets*2;i++)
		result[i]=0;


/* CUDA result */
int *d_result;
int size_result = (NTargets * 2) * sizeof(int);
hipMalloc((void**) &d_result, size_result);
hipMemcpy(d_result, result, size_result, hipMemcpyHostToDevice);


t= new tweet[Ntweets];

/* CUDA tweets */
tweet *d_tweets;
int size_tweets = Ntweets * sizeof(tweet);
hipMalloc((void**) &d_tweets, size_tweets);
hipMemcpy( d_tweets, t, size_tweets, hipMemcpyHostToDevice);


	//Execute
	while(!feof(in))
	{
		for(j=0;j< Ntweets; j++)
			t[j].load_from_file(in);

		/* CALL CUDA KERNEL */
		kernel<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>> (d_dict, Ntargets, candidate_tag, Ntweets, d_tweets, d_result);
	}

	/* CUDA copy result */
	hipMemcpy(result, d_result, sizeof( int ) , hipMemcpyDeviceToHost );


	/* CUDA free */
	hipFree(d_dict);
	hipFree(d_result);
	hipFree(d_tweets);


//print results:
	for (i = 0; i < Ntargets; i++)
		printf("Candidate: %d\n\tPositive: %d\tNegative: %d\n", i, result[i * 2], result[i * 2+1]);
	delete(dict);
	for (i = 0; i < Ntargets; i++)
		delete(candidate_tag[i]);
	delete(candidate_tag);
	fclose(in);	
	printf ("\nExecution time: %.3f\n",((float)clock()/CLOCKS_PER_SEC) - timestart);
    
	puts("Press any key to exit...");
	char c[1];
	gets(c);
	return EXIT_SUCCESS;
}
